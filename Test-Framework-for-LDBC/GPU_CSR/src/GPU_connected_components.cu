#include "hip/hip_runtime.h"
#include <GPU_connected_components.cuh>

/*template std::vector<std::vector<int>> gpu_connected_components<int>(CSR_graph<int>&);
template std::vector<std::vector<int>> gpu_connected_components<float>(CSR_graph<float>&);
template std::vector<std::vector<int>> gpu_connected_components<double>(CSR_graph<double>&);
template std::vector<std::vector<int>> gpu_connected_components<long long>(CSR_graph<long long>&);*/

__device__ int findRoot(int* parent, int i) {
    //Recursively searching for the ancestor of node i
    while (i != parent[i])
        i = parent[i];//
    return i;
}

__global__ void Hook(int* parent, int* Start_v, int* End_v, int E) {
    //Merge operations on each edge
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    //Calculate thread ID
    if (id < E) {
		int u = Start_v[id];
		int v = End_v[id];
        //u,v are the starting and ending points of the edge
        int rootU = findRoot(parent, u);
        int rootV = findRoot(parent, v);
        //Obtain Root Node
        while (rootU != rootV) {
            int expected = rootU > rootV ? rootU : rootV;
            int desired = rootU < rootV ? rootU : rootV;
           //During multi-core operations, the root node may be manipulated by other threads, so locking is necessary for the operation
            int observed = atomicCAS(&parent[expected], expected, desired);
            /*
            compare and swap
            int atomicCAS(int* address, int compare, int val);
            Check if the address and compare are the same. If they are the same, enter address as desired. Otherwise, no action will be taken
            observed = parent[expected]

            */
            
          
            if (observed == expected)//If the observed values are correct and the merge operation is successful, exit the loop
                break;
            //If the observed value has been modified, the modified new root node needs to be obtained
            rootU = findRoot(parent, u);
            rootV = findRoot(parent, v);
        }
    }
}

//template <typename T>
std::vector<std::vector<int>> gpu_connected_components(CSR_graph<double>& input_graph, float* elapsedTime) {
    int N = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
    int E = input_graph.OUTs_Edges.size();
    //Number of nodes and edges
    
    int* Start_v;
    int* End_v;
    int* Parent;
    // Allocate GPU memory
    hipMallocManaged((void**)&Start_v, E * sizeof(int));
    memset(Start_v, 0, E * sizeof(int));
    hipMallocManaged((void**)&End_v, E * sizeof(int));
    memset(End_v, 0, E * sizeof(int));
    hipMallocManaged((void**)&Parent, N * sizeof(int));
    //Forming an edge list
    // Copy data to GPU
    for (int i = 0; i < N; i++) {
        for (int j = input_graph.OUTs_Neighbor_start_pointers[i]; j < input_graph.OUTs_Neighbor_start_pointers[i + 1]; j++) {
			Start_v[j] = i;
			End_v[j] = input_graph.OUTs_Edges[j];
		}
        Parent[i] = i;//initialization
    }

    hipEvent_t start, stop;
    //Used to create events and measure the time of GPU operations.
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = 0;
    //Disperse E operations on threads
    blocksPerGrid = (E + threadsPerBlock - 1) / threadsPerBlock;
    Hook<<<blocksPerGrid, threadsPerBlock>>>(Parent, Start_v, End_v, E);
    hipDeviceSynchronize();
    hipError_t cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_status));
		return std::vector<std::vector<int>>();
	}
    // Process components on CPU
    std::vector<std::vector<int>> components;
    std::vector<std::vector<int>> componentLists(N);
    //Using a linked list to record connected components
    for (int i = 0; i < N; i++) {
        if (Parent[i] != i) {
            //If it is not the root node, add the node to the linked list of the root node it belongs to
            int j = i;
            while (Parent[j] != j)
                j = Parent[j];
            Parent[i] = j;
            componentLists[j].push_back(i);
        }
        else  //The root node is directly added to the root node linked list
            componentLists[i].push_back(i);
    }

    for (int i = 0; i < N; i++) {
		if (componentLists[i].size() > 0)
            //Filter non empty connected components
			components.push_back(componentLists[i]);
	}

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsedTime, start, stop);
    //printf("Cost time is %f\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free GPU memory
    hipFree(Start_v);
    hipFree(End_v);
    hipFree(Parent);

    return components;
}

/*int main()
{
    graph_v_of_v<int> graph;
    graph.txt_read("example_graph.txt");
    ARRAY_graph<int> arr_graph = graph.toARRAY();
    float sum = 0;
    int it_cnt = 100;
    for (int i = 0; i < it_cnt; i++) {
        gpu_connected_components<int>(arr_graph);
        if (i > 0)
            sum += elapsedTime;
        elapsedTime = 0;
    }
    printf("average cost time is %f ms\n", sum / it_cnt);
    return 0;
}*/

/*

nvcc -O3 -std=c++17 -o Union-Find.out Union-Find.cu
./Union-Find.out
rm Union-Find.out

*/
