#include "hip/hip_runtime.h"
#include <GPU_shortest_paths.cuh>

__device__ __forceinline__ double atomicMinDouble (double * addr, double value) {
    double old;
    old = __longlong_as_double(atomicMin((long long *)addr, __double_as_longlong(value)));
    return old;
}

__global__ void Relax(int* out_pointer, int* out_edge, double* out_edge_weight, double* dis, int* queue, int* queue_size, int* visited) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < *queue_size) {
        int v = queue[idx];

        for (int i = out_pointer[v]; i < out_pointer[v + 1]; i++) {
            int new_v = out_edge[i];
            double weight = out_edge_weight[i];

            double new_w = dis[v] + weight;

            double old = atomicMinDouble(&dis[new_v], new_w);

            if (old <= new_w)
				continue;

            atomicExch(&visited[new_v], 1);
        }
    }
}

__global__ void CompactQueue(int V, int* next_queue, int* next_queue_size, int* visited) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < V && visited[idx]) {
        int pos = atomicAdd(next_queue_size, 1);
        next_queue[pos] = idx;
        visited[idx] = 0;
    }
}

void gpu_shortest_paths(CSR_graph<double>& input_graph, int source, std::vector<double>& distance, float* elapsedTime, double max_dis) {
    int V = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
    int E = input_graph.OUTs_Edges.size();

    double* dis;
    int* out_edge = input_graph.out_edge;
    double* out_edge_weight = input_graph.out_edge_weight;
    int* out_pointer = input_graph.out_pointer;
    int* visited;
    
    int* queue, * next_queue;
    int* queue_size, * next_queue_size;

    hipMallocManaged((void**)&dis, V * sizeof(double));
    hipMallocManaged((void**)&visited, V * sizeof(int));
    hipMallocManaged((void**)&queue, V * sizeof(int));
    hipMallocManaged((void**)&next_queue, V * sizeof(int));
    hipMallocManaged((void**)&queue_size, sizeof(int));
    hipMallocManaged((void**)&next_queue_size, sizeof(int));

    for (int i = 0; i < V; i++) {
		dis[i] = max_dis;
		visited[i] = 0;
	}
    dis[source] = 0;


    *queue_size = 1;
    queue[0] = source;
    *next_queue_size = 0;

    int threadsPerBlock = 1024;
    int numBlocks = 0;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    while (*queue_size > 0) {
		numBlocks = (*queue_size + threadsPerBlock - 1) / threadsPerBlock;
		Relax <<< numBlocks, threadsPerBlock >>> (out_pointer, out_edge, out_edge_weight, dis, queue, queue_size, visited);
		hipDeviceSynchronize();

        hipError_t cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            fprintf(stderr, "Relax kernel launch failed: %s\n", hipGetErrorString(cuda_status));
            return;
        }

		numBlocks = (V + threadsPerBlock - 1) / threadsPerBlock;
		CompactQueue <<< numBlocks, threadsPerBlock >>> (V, next_queue, next_queue_size, visited);
		hipDeviceSynchronize();

        cuda_status = hipGetLastError();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "CompactQueue kernel launch failed: %s\n", hipGetErrorString(cuda_status));
			return;
		}
		
        std::swap(queue, next_queue);

		*queue_size = *next_queue_size;
        *next_queue_size = 0;
	}

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(distance.data(), dis, V * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dis);
    hipFree(visited);
    hipFree(queue);
    hipFree(next_queue);
    hipFree(queue_size);
    hipFree(next_queue_size);

    return;
}

std::map<long long int, double> getGPUSSSP(LDBC<double> & graph, CSR_graph<double> & csr_graph){
    std::vector<double> gpuSSSPvec(graph.V, 0);
    gpu_shortest_paths(csr_graph, graph.sssp_src, gpuSSSPvec, 0, 10000000000);

    std::map<long long int,   double> strId2value;

    std::vector<long long int> converted_numbers;

    for (const auto& str : graph.vertex_id_to_str) {
        long long int num = std::stoll(str);
        converted_numbers.push_back(num);
    }

    std::sort(converted_numbers.begin(), converted_numbers.end());

	for( int i = 0; i < gpuSSSPvec.size(); ++i){
		strId2value.emplace(converted_numbers[i], gpuSSSPvec[i]);
    }

	// std::string path = "../data/cpu_bfs_75.txt";
	// storeResult(strId2value, path);//ldbc file

    return strId2value;
}

std::vector<std::string> gpu_shortest_paths_v2(LDBC<double> & graph, CSR_graph<double> &csr_graph){
    std::vector<double> gpuSSSPvec(graph.V, 0);
    gpu_shortest_paths(csr_graph, graph.sssp_src, gpuSSSPvec, 0, 10000000000);

    std::vector<std::string> resultVec;

    for(auto & it : gpuSSSPvec){
		resultVec.push_back(std::to_string(it));
	}

	return resultVec;
}