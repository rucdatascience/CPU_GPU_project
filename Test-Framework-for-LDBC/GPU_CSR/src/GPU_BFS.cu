#include "hip/hip_runtime.h"
// #include <GPU_BFS.cuh>
#include "../header/GPU_BFS.cuh"
//It's not that the CPU tasks are assigned to the GPU, but rather that the GPU determines which part of the task to complete based on its own ID number
__global__ void bfs_kernel(int* edges, int* start, int* visited, int* queue, int* next_queue, int* queue_size, int* next_queue_size, int max_depth) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
   // Grid is divided into 1 dimension, Block is divided into 1 dimension.
    if (tid < *queue_size) {
        int vertex = queue[tid];
        int depth = visited[vertex];
        for (int edge = start[vertex]; edge < start[vertex + 1]; edge++) {
            //The traversal range of node edges is given by the start array
//Traverse adjacent edges
            int neighbor = edges[edge];
            if (visited[neighbor] >= max_depth && depth < max_depth) {
                visited[neighbor] = depth + 1;
                int pos = atomicAdd(next_queue_size, 1);//AtomicAdd is an atomic addition function in CUDA, used to ensure data consistency and correctness when multiple threads modify the same global variable simultaneously.
                next_queue[pos] = neighbor;//Generate the next queue, which could be understood as a queue joining operation
            }
        }
    }
}

//template <typename T>
std::vector<int> cuda_bfs(CSR_graph<double>& input_graph, int source_vertex, float* elapsedTime, int max_depth) {
    int V = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
    int E = input_graph.OUTs_Edges.size();
    
    std::vector<int> depth(V, max_depth);

    if (source_vertex < 0 || source_vertex >= V) {
        fprintf(stderr, "Invalid source vertex\n");
        return depth;
    }

    int* visited;
    int* queue, * next_queue;
    int* queue_size, * next_queue_size;

    int *edges = input_graph.out_edge, *start = input_graph.out_pointer;
    
    //Allocate GPU memory
    hipMallocManaged((void**)&visited, V * sizeof(int));
    hipMallocManaged((void**)&queue, V * sizeof(int));
    hipMallocManaged((void**)&next_queue, V * sizeof(int));
    hipMallocManaged((void**)&queue_size, sizeof(int));
    hipMallocManaged((void**)&next_queue_size, sizeof(int));
    //Transferring the read in data to the GPU
   /*  hipMemcpy(edges, input_graph.OUTs_Edges.data(), E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(start, input_graph.OUTs_Neighbor_start_pointers.data(), (V+1) * sizeof(int), hipMemcpyHostToDevice); // should be V+1???????
     */
    queue[0] = source_vertex;
    for (int i = 0; i < V; i++)
        visited[i] = max_depth;//vector initialization
    visited[source_vertex] = 0;//Record whether the node has been accessed
    *queue_size = 1, *next_queue_size = 1;

    int threadsPerBlock = 1024;
    int numBlocks = 0;

    hipEvent_t start_clock, stop_clock;

    hipEventCreate(&start_clock);
    hipEventCreate(&stop_clock);
    hipEventRecord(start_clock, 0);

    while (*queue_size > 0) {
        //The BFS operation continues to loop until the queue is empty
        numBlocks = (*queue_size + threadsPerBlock - 1) / threadsPerBlock;
        //Assign tasks to threads
        bfs_kernel << <numBlocks, threadsPerBlock >> > (edges, start, visited, queue, next_queue, queue_size, next_queue_size, max_depth);
        hipDeviceSynchronize();
        //Print error messages
        hipError_t cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_status));
            return depth;
        }
        //Exchange to obtain the queue for the next round of circulation
        std::swap(queue, next_queue);
        *queue_size = *next_queue_size;
        *next_queue_size = 0;
    }
    //accord time cost
    hipEventRecord(stop_clock, 0);
    hipEventSynchronize(stop_clock);
    hipEventElapsedTime(elapsedTime, start_clock, stop_clock);

    hipEventDestroy(start_clock);
    hipEventDestroy(stop_clock);
    //free memory
    hipMemcpy(depth.data(), visited, V * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(visited);
    hipFree(queue);
    hipFree(next_queue);
    hipFree(queue_size);
    hipFree(next_queue_size);


    return depth;
}

/*int main()
{
    std::string file_path;
    std::cout << "Please input the file path of the graph: ";
    std::cin >> file_path;
    graph_v_of_v<int> graph;
    graph.txt_read(file_path);
    ARRAY_graph<int> array_graph = graph.toARRAY();
    int V = array_graph.Neighbor_start_pointers.size();
    cuda_bfs(array_graph, 0);
    float sum = 0;
    for (int i = 0; i < V; i++) {
        cuda_bfs(array_graph, i);
        sum += elapsedTime;
        elapsedTime = 0;
    }
    printf("GPU average cost time: %f ms\n", sum / V);
    return 0;
}*/

/*

nvcc -O3 -std=c++17 -o GPU_BFS.out GPU_BFS.cu
./GPU_BFS.out
rm GPU_BFS.out

*/
