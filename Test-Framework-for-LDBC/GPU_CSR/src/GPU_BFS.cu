#include "hip/hip_runtime.h"
#include <GPU_BFS.cuh>

__global__ void bfs_Relax(int* start, int* edge, int* depth, int* visited, int* queue, int* queue_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < *queue_size) {
        int v = queue[idx];

        for (int i = start[v]; i < start[v + 1]; i++) {
            int new_v = edge[i];

            int new_depth = depth[v] + 1;

            int old = atomicMin(&depth[new_v], new_depth);

            if (old <= new_depth)
				continue;

            atomicExch(&visited[new_v], 1);
        }
    }
}

__global__ void bfs_CompactQueue(int V, int* next_queue, int* next_queue_size, int* visited) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < V && visited[idx]) {
        int pos = atomicAdd(next_queue_size, 1);
        next_queue[pos] = idx;
        visited[idx] = 0;
    }
}

//It's not that the CPU tasks are assigned to the GPU, but rather that the GPU determines which part of the task to complete based on its own ID number
__global__ void bfs_kernel(int* edges, int* start, int* visited, int* queue, int* next_queue, int* queue_size, int* next_queue_size, int max_depth) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
   // Grid is divided into 1 dimension, Block is divided into 1 dimension.
    if (tid < *queue_size) {
        int vertex = queue[tid];
        int depth = visited[vertex];
        for (int edge = start[vertex]; edge < start[vertex + 1]; edge++) {
            //The traversal range of node edges is given by the start array
//Traverse adjacent edges
            int neighbor = edges[edge];
            if (visited[neighbor] >= max_depth && depth < max_depth) {
                visited[neighbor] = depth + 1;
                int pos = atomicAdd(next_queue_size, 1);//AtomicAdd is an atomic addition function in CUDA, used to ensure data consistency and correctness when multiple threads modify the same global variable simultaneously.
                next_queue[pos] = neighbor;//Generate the next queue, which could be understood as a queue joining operation
            }
        }
    }
}

//template <typename T>
std::vector<int> cuda_bfs(CSR_graph<double>& input_graph, int source, int max_depth) {
    int V = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
    int E = input_graph.OUTs_Edges.size();

    int* depth;
    int* edge = input_graph.out_edge;

    int* start = input_graph.out_pointer;
    int* visited;
    
    int* queue, * next_queue;
    int* queue_size, * next_queue_size;

    hipMallocManaged((void**)&depth, V * sizeof(int));
    hipMallocManaged((void**)&visited, V * sizeof(int));
    hipMallocManaged((void**)&queue, V * sizeof(int));
    hipMallocManaged((void**)&next_queue, V * sizeof(int));
    hipMallocManaged((void**)&queue_size, sizeof(int));
    hipMallocManaged((void**)&next_queue_size, sizeof(int));

    for (int i = 0; i < V; i++) {
		depth[i] = max_depth;
		visited[i] = 0;
	}
    depth[source] = 0;


    *queue_size = 1;
    queue[0] = source;
    *next_queue_size = 0;

    int threadsPerBlock = 1024;
    int numBlocks = 0;

    std::vector<int> res(V, max_depth);

    while (*queue_size > 0) {
		numBlocks = (*queue_size + threadsPerBlock - 1) / threadsPerBlock;
		bfs_Relax <<< numBlocks, threadsPerBlock >>> (start, edge, depth, visited, queue, queue_size);
		hipDeviceSynchronize();

        hipError_t cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            fprintf(stderr, "Relax kernel launch failed: %s\n", hipGetErrorString(cuda_status));
            return res;
        }

		numBlocks = (V + threadsPerBlock - 1) / threadsPerBlock;
		bfs_CompactQueue <<< numBlocks, threadsPerBlock >>> (V, next_queue, next_queue_size, visited);
		hipDeviceSynchronize();

        cuda_status = hipGetLastError();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "CompactQueue kernel launch failed: %s\n", hipGetErrorString(cuda_status));
			return res;
		}
		
        std::swap(queue, next_queue);

		*queue_size = *next_queue_size;
        *next_queue_size = 0;
	}

    hipMemcpy(res.data(), depth, V * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(depth);
    hipFree(visited);
    hipFree(queue);
    hipFree(next_queue);
    hipFree(queue_size);
    hipFree(next_queue_size);

    return res;
}

std::map<long long int, int> getGPUBFS(graph_structure<double> & graph, CSR_graph<double> &csr_graph){
    std::vector<int> gpuBfsVec = cuda_bfs(csr_graph, graph.bfs_src, 0);
    
    std::map<long long int,   int> strId2value;

    std::vector<long long int> converted_numbers;

    for (const auto& str : graph.vertex_id_to_str) {
        long long int num = std::stoll(str);
        converted_numbers.push_back(num);
    }

    std::sort(converted_numbers.begin(), converted_numbers.end());

	for( int i = 0; i < gpuBfsVec.size(); ++i){
		strId2value.emplace(converted_numbers[i], gpuBfsVec[i]);
    }

	// std::string path = "../data/cpu_bfs_75.txt";
	// storeResult(strId2value, path);//ldbc file

    return strId2value;
}

std::vector<std::string> cuda_bfs_v2(graph_structure<double> & graph, CSR_graph<double> &csr_graph){
    std::vector<int> gpuBfsVec = cuda_bfs(csr_graph, graph.bfs_src);

    std::vector<std::string> resultVec;

    for(auto & it : gpuBfsVec){
		resultVec.push_back(std::to_string(it));
	}

	return resultVec;
}

std::vector<std::pair<std::string, int>> Cuda_Bfs(graph_structure<double>& graph, CSR_graph<double>& csr_graph, std::string src_v, int min_depth, int max_depth) {
    int src_v_id = graph.vertex_str_to_id[src_v];
    std::vector<int> gpuBfsVec = cuda_bfs(csr_graph, src_v_id, max_depth);

    return graph.res_trans_id_val(gpuBfsVec);
}
