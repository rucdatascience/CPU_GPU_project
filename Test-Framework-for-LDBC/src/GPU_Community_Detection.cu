#include "hip/hip_runtime.h"
#include <GPU_Community_Detection.cuh>
using namespace std;

static int CD_GRAPHSIZE;
static int CD_ITERATION;
static vector<int> outs_ptr,ins_ptr, outs_neighbor,ins_neighbor;

static int* outs_ptr_gpu,*ins_ptr_gpu;
static int* labels_gpu, * outs_neighbor_gpu,*ins_neighbor_gpu;
static int* reduce_label, * reduce_label_count;


template <typename T>
void make_csr(graph_structure<T> &graph, int& CD_GRAPHSIZE)
{
   
    CSR_graph<T> ARRAY_graph;
    ARRAY_graph=graph.toCSR();

    CD_GRAPHSIZE = ARRAY_graph.OUTs_Neighbor_start_pointers.size() - 1;

    outs_ptr.resize(CD_GRAPHSIZE + 1);
    outs_ptr=ARRAY_graph.OUTs_Neighbor_start_pointers;
    ins_ptr.resize(CD_GRAPHSIZE + 1);
    ins_ptr=ARRAY_graph.INs_Neighbor_start_pointers;

    outs_neighbor=ARRAY_graph.OUTs_Edges;
    ins_neighbor=ARRAY_graph.INs_Edges;

}


__global__ void init_label(int* labels_gpu,int CD_GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 0 && tid < CD_GRAPHSIZE)
    {
        labels_gpu[tid] = tid;
    }
}

__global__ void LPA(int* outs_ptr_gpu, int* labels_gpu, int* outs_neighbor_gpu, int* reduce_label, int* reduce_label_count,int CD_GRAPHSIZE,int BLOCK_PER_VER,int * ins_ptr_gpu,int* ins_neighbor_gpu,int epoch_it,int epoch_size)
{
    extern __shared__ int shared_memory[];
    int* label_counts = shared_memory;
    int* label = (int*)&label_counts[blockDim.x];

    int block_order=blockIdx.x;
    int ver = block_order / BLOCK_PER_VER+epoch_it*epoch_size;
    if(ver>=CD_GRAPHSIZE) return;
    int segment_order = block_order % BLOCK_PER_VER;
    int tid = (segment_order) * blockDim.x + threadIdx.x;

    int stid = threadIdx.x;
    if (tid == ver)
    {
        label_counts[stid] = 1;
    }
    else
    {
        label_counts[stid] = 0;
    }
    label[stid] = tid;

    __syncthreads();

    int outs_start = outs_ptr_gpu[ver], outs_end = outs_ptr_gpu[ver + 1];
    if (tid < outs_end - outs_start)
    {
        int neighbor_label = labels_gpu[outs_neighbor_gpu[outs_start + tid]];
        if (neighbor_label >= segment_order * CD_THREAD_PER_BLOCK && neighbor_label < (segment_order + 1) * CD_THREAD_PER_BLOCK)
            atomicAdd(&label_counts[neighbor_label - segment_order * CD_THREAD_PER_BLOCK], 1);
    }
    int ins_start = ins_ptr_gpu[ver], ins_end = ins_ptr_gpu[ver + 1];
    if (tid < ins_end - ins_start)
    {
        int neighbor_label = labels_gpu[ins_neighbor_gpu[ins_start + tid]];
        if (neighbor_label >= segment_order * CD_THREAD_PER_BLOCK && neighbor_label < (segment_order + 1) * CD_THREAD_PER_BLOCK)
            atomicAdd(&label_counts[neighbor_label - segment_order * CD_THREAD_PER_BLOCK], 1);
    }


    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (stid < s)
        {
            if (label_counts[stid] < label_counts[stid + s])
            {
                label_counts[stid] = label_counts[stid + s];
                label[stid] = label_counts[stid + s];
            }
            else if (label_counts[stid] == label_counts[stid + s] && label[stid] > label_counts[stid + s])
            {
                label[stid] = label_counts[stid + s];
            }
        }
        __syncthreads();
    }
    if(stid==0){
        reduce_label_count[block_order] = label_counts[0];
        reduce_label[block_order] = label[0];
    }
    
    return;
}

__global__ void Updating_label(int* reduce_label, int* reduce_label_count,  int* labels_gpu,int CD_GRAPHSIZE,int BLOCK_PER_VER,int epoch_it,int epoch_size)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >=epoch_size)
        return;
    int ver=tid+epoch_it*epoch_size;

    int cont = 1, label = labels_gpu[ver];

    int start = tid * BLOCK_PER_VER, end = start + BLOCK_PER_VER;
    for (int i = start; i < end; ++i)
    {
        if (reduce_label_count[i] > cont)
        {
            cont = reduce_label_count[i];
            label = reduce_label[i];
        }
        else if (reduce_label_count[i] == cont && reduce_label[i] < label)
        {
            label = reduce_label[i];
        }
    }

    labels_gpu[ver] = label;
    return;
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << "Error: " << msg << " (" << hipGetErrorString(err) << ")" << endl;
        exit(EXIT_FAILURE);
    }
}

void checkDeviceProperties() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cout << "Device name: " << prop.name << endl;
    cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
    cout << "Max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << endl;
    cout << "Max blocks per dimension: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << endl;
    cout << "Max shared memory per block: " << prop.sharedMemPerBlock << " bytes" << endl;
    cout << "Total global memory: " << prop.totalGlobalMem << " bytes" << endl;
}

int Community_Detection(graph_structure<double>& graph, float* elapsedTime)
{
    make_csr(graph,CD_GRAPHSIZE);
    CD_ITERATION=graph.cdlp_max_its;
    int BLOCK_PER_VER=((CD_GRAPHSIZE + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK);
    int set_block=1e9;
    int epoch_size=set_block/BLOCK_PER_VER;
    int epoch_iteration=(CD_GRAPHSIZE+epoch_size-1)/epoch_size;
    int REDUCE_BLOCK_PER_GRID=(epoch_size + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK;
    
    cout<<"epoch_size : "<<epoch_size<<endl;
    cout<<"epoch_iteration : "<<epoch_iteration<<endl;
    cout<<"BLOCK_PER_VER : "<<BLOCK_PER_VER<<"  reduce_size : "<<(CD_GRAPHSIZE) * BLOCK_PER_VER<<endl;
    dim3 blockPerGrid((CD_GRAPHSIZE + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK, 1, 1);
    dim3 useBlock(epoch_size*BLOCK_PER_VER, 1, 1);

    dim3 threadPerBlock(CD_THREAD_PER_BLOCK, 1, 1);
    dim3 reduceBlock(REDUCE_BLOCK_PER_GRID, 1, 1);

    hipMalloc(&outs_ptr_gpu, (CD_GRAPHSIZE + 1) * sizeof(int));
    hipMalloc(&ins_ptr_gpu, (CD_GRAPHSIZE + 1) * sizeof(int));
    hipMalloc(&labels_gpu, CD_GRAPHSIZE * sizeof(int));
    hipMalloc(&outs_neighbor_gpu, outs_neighbor.size() * sizeof(int));
    hipMalloc(&ins_neighbor_gpu, ins_neighbor.size() * sizeof(int));
    hipMalloc(&reduce_label, set_block * sizeof(int));
    hipMalloc(&reduce_label_count, set_block * sizeof(int));
    hipMemcpy(outs_ptr_gpu, outs_ptr.data(), outs_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ins_ptr_gpu, ins_ptr.data(), ins_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(outs_neighbor_gpu, outs_neighbor.data(), outs_neighbor.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ins_neighbor_gpu, ins_neighbor.data(), ins_neighbor.size() * sizeof(int), hipMemcpyHostToDevice);

    checkDeviceProperties();
    int it=0;

    init_label << <blockPerGrid, threadPerBlock >> > (labels_gpu,CD_GRAPHSIZE);
    hipDeviceSynchronize();

    hipEvent_t GPUstart, GPUstop;
    hipEventCreate(&GPUstart);
    hipEventCreate(&GPUstop);
    hipEventRecord(GPUstart, 0);
    hipError_t err;
     while (it < CD_ITERATION) {
        cout << "iteration : " << it << endl;
        it++;
        for(int i=0;i<epoch_iteration;i++){
            LPA <<<useBlock, threadPerBlock, sizeof(int) * CD_THREAD_PER_BLOCK * 2>>> (
                outs_ptr_gpu, labels_gpu, outs_neighbor_gpu, reduce_label, reduce_label_count, CD_GRAPHSIZE, BLOCK_PER_VER, ins_ptr_gpu, ins_neighbor_gpu,i,epoch_size);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after LPA");

            Updating_label <<<reduceBlock, threadPerBlock>>> (
                reduce_label, reduce_label_count, labels_gpu, CD_GRAPHSIZE, BLOCK_PER_VER,i,epoch_size);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after Updating_label");
        }
        

        
    }

    hipEventRecord(GPUstop, 0);
    hipEventSynchronize(GPUstop);


    hipEventElapsedTime(elapsedTime, GPUstart, GPUstop);
    hipEventDestroy(GPUstart);
    hipEventDestroy(GPUstop);

    hipFree(outs_ptr_gpu);
    hipFree(labels_gpu);
    hipFree(outs_neighbor_gpu);
    hipFree(reduce_label);
    hipFree(reduce_label_count);

    return 0;
}
