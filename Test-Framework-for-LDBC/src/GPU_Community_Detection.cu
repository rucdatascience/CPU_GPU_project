#include "hip/hip_runtime.h"
#include <GPU_Community_Detection.cuh>
using namespace std;

static int CD_GRAPHSIZE;
static int CD_ITERATION;
static int CD_SET_THREAD;
// static int CD_M;
static vector<int> outs_ptr, ins_ptr, outs_neighbor, ins_neighbor, in_out_ptr;

static int *in_out_ptr_gpu;
static int *outs_ptr_gpu, *ins_ptr_gpu;
static int *outs_neighbor_gpu, *ins_neighbor_gpu;
static int *new_labels_gpu, *labels_gpu;
static int *global_space_for_label;

template <typename T>
void pre_set(graph_structure<T> &graph, int &CD_GRAPHSIZE)
{

    CSR_graph<T> ARRAY_graph;
    ARRAY_graph = graph.toCSR();

    CD_ITERATION = graph.cdlp_max_its;
    CD_GRAPHSIZE = ARRAY_graph.OUTs_Neighbor_start_pointers.size() - 1;
    // CD_SET_THREAD = 100;

    outs_ptr.resize(CD_GRAPHSIZE + 1);
    outs_ptr = ARRAY_graph.OUTs_Neighbor_start_pointers;
    ins_ptr.resize(CD_GRAPHSIZE + 1);
    ins_ptr = ARRAY_graph.INs_Neighbor_start_pointers;
    in_out_ptr.resize(CD_GRAPHSIZE + 1);
    in_out_ptr[0] = 0;
    for (int i = 1; i <= CD_GRAPHSIZE; ++i)
    {
        in_out_ptr[i] = in_out_ptr[i - 1] + (ins_ptr[i] - ins_ptr[i - 1]) + (outs_ptr[i] - outs_ptr[i - 1])+1;
    }
    // for(int i=0;i<10000;++i){
    //     cout<<outs_ptr[i]<<"  "<<ins_ptr[i]<<"  "<<in_out_ptr[i]<<endl;
    // }
    outs_neighbor = ARRAY_graph.OUTs_Edges;
    ins_neighbor = ARRAY_graph.INs_Edges;

    // int sum = 0;
    // int max = 0;
    // for (int i = 0; i < CD_GRAPHSIZE; ++i)
    // {
    //     int cont = 0;
    //     cont += (ins_ptr[i + 1] - ins_ptr[i] + outs_ptr[i + 1] - outs_ptr[i]);
    //     if (cont > max)
    //     {
    //         max = cont;
    //     }
    //     sum += cont;
    // }
    // cout << "max degree : " << max << endl;
    // cout << "avg degree : " << sum / CD_GRAPHSIZE << endl;
    // int t = (size_t)(20LL * (1LL << 30)) / (max * 2 * 4);
    // cout << "use 20GB for max degree : " << t << endl;

    // CD_M = max;
    // CD_SET_THREAD = t > 10000 ? 10000 : t;
}

__global__ void init_label(int *labels_gpu, int CD_GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 0 && tid < CD_GRAPHSIZE)
    {
        labels_gpu[tid] = tid;
    }
}

__global__ void extract_labels(int *in_out_ptr_gpu, int *ins_ptr_gpu, int *outs_ptr_gpu, int *ins_neighbor_gpu, int *outs_neighbor_gpu, int *labels_gpu, int *labels_out, int CD_GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= CD_GRAPHSIZE)
        return;

    int start = in_out_ptr_gpu[tid];
    int len_out = outs_ptr_gpu[tid + 1] - outs_ptr_gpu[tid];
    int len_in = ins_ptr_gpu[tid + 1] - ins_ptr_gpu[tid];
    // int end = in_out_ptr_gpu[tid + 1];
    for (int i = 0; i < len_out; ++i)
    {
        labels_out[start + i] = labels_gpu[outs_neighbor_gpu[outs_ptr_gpu[tid] + i]];
    }
    for (int i = 0; i < len_in; ++i)
    {
        labels_out[start + len_out + i] = labels_gpu[ins_neighbor_gpu[ins_ptr_gpu[tid] + i]];
    }
    labels_out[start + len_out + len_in] = labels_gpu[tid];
    return;
}

__global__ void parallel_sort_labels(int *in_out_ptr_gpu, int *labels_out, int CD_GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= CD_GRAPHSIZE)
        return;

    int start = in_out_ptr_gpu[tid];
    int end = in_out_ptr_gpu[tid + 1];
    thrust::sort(thrust::device, labels_out + start, labels_out + end);
}

void checkCudaError(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        cerr << "Error: " << msg << " (" << hipGetErrorString(err) << ")" << endl;
        exit(EXIT_FAILURE);
    }
}

void checkDeviceProperties()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cout << "Device name: " << prop.name << endl;
    cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
    cout << "Max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << endl;
    cout << "Max blocks per dimension: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << endl;
    cout << "Max shared memory per block: " << prop.sharedMemPerBlock << " bytes" << endl;
    cout << "Total global memory: " << prop.totalGlobalMem << " bytes" << endl;
}

void get_size()
{
    size_t freeMem = 0;
    size_t totalMem = 0;

    hipError_t err = hipMemGetInfo(&freeMem, &totalMem);

    if (err != hipSuccess)
    {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cout << "Free memory: " << freeMem << " Byte" << std::endl;
    std::cout << "Total memory: " << totalMem << " Byte" << std::endl;
    size_t a = CD_GRAPHSIZE * 8;
    cout << "space for single thread  : " << a << endl;
    cout << "max thread num : " << freeMem / a << endl;
    size_t t = (size_t)(20LL * (1LL << 30)) / a;

    cout << "if use 20GB : " << t << endl;

    return;
}

__global__ void LPA(int *global_space_for_label, int *in_out_ptr_gpu, int *labels_gpu, int *new_labels_gpu, int CD_GRAPHSIZE)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= CD_GRAPHSIZE)
        return;
    int start = in_out_ptr_gpu[tid], end = in_out_ptr_gpu[tid + 1];
    int current_label = -1;
    int current_count = 0;
    int max_label = current_label;
    int max_count = current_count;
    for (int i = start; i < end; ++i)
    {
        if (global_space_for_label[i] == current_label)
        {
            current_count++;
        }
        else
        {
            if (current_count > max_count)
            {
                max_count = current_count;
                max_label = current_label;
            }
            else if (current_count == max_count && current_label < max_label)
            {
                max_label = current_label;
            }
            current_label = global_space_for_label[i];
            current_count = 1;
        }
    }
    if (current_count > max_count)
    {
        max_count = current_count;
        max_label = current_label;
    }
    else if (current_count == max_count && current_label < max_label)
    {
        max_label = current_label;
    }
    new_labels_gpu[tid] = max_label;
}

int Community_Detection(graph_structure<double> &graph, float *elapsedTime, vector<int> &ans)
{
    pre_set(graph, CD_GRAPHSIZE);

    dim3 init_label_block((CD_GRAPHSIZE + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK, 1, 1);
    dim3 init_label_thread(CD_THREAD_PER_BLOCK, 1, 1);
    // dim3 LPA_block((CD_SET_THREAD + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK, 1, 1);
    // dim3 LPA_thread(CD_THREAD_PER_BLOCK, 1, 1);

    // cout << 1 << endl;
    hipMallocManaged(&outs_ptr_gpu, (CD_GRAPHSIZE + 1) * sizeof(int));
    hipMallocManaged(&ins_ptr_gpu, (CD_GRAPHSIZE + 1) * sizeof(int));
    hipMallocManaged(&labels_gpu, CD_GRAPHSIZE * sizeof(int));
    hipMallocManaged(&new_labels_gpu, CD_GRAPHSIZE * sizeof(int));
    hipMalloc(&outs_neighbor_gpu, outs_neighbor.size() * sizeof(int));
    hipMalloc(&ins_neighbor_gpu, ins_neighbor.size() * sizeof(int));
    hipMallocManaged(&global_space_for_label, (outs_neighbor.size() + ins_neighbor.size() + CD_GRAPHSIZE) * sizeof(int));
    hipMallocManaged(&in_out_ptr_gpu, (CD_GRAPHSIZE + 1) * sizeof(int));

    hipMemcpy(outs_ptr_gpu, outs_ptr.data(), outs_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ins_ptr_gpu, ins_ptr.data(), ins_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(outs_neighbor_gpu, outs_neighbor.data(), outs_neighbor.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ins_neighbor_gpu, ins_neighbor.data(), ins_neighbor.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(in_out_ptr_gpu, in_out_ptr.data(), in_out_ptr.size() * sizeof(int), hipMemcpyHostToDevice);

    // checkDeviceProperties();
    // get_size();
    // cout << 2 << endl;
    hipError_t err;
    init_label<<<init_label_block, init_label_thread>>>(labels_gpu, CD_GRAPHSIZE);
    err = hipDeviceSynchronize();
    checkCudaError(err, "hipDeviceSynchronize after init_label");
    // cout << 3 << endl;

    hipEvent_t GPUstart, GPUstop;
    hipEventCreate(&GPUstart);
    hipEventCreate(&GPUstop);
    hipEventRecord(GPUstart, 0);

    int it = 0;
    // cout << 4 << endl;
    // cout << "total epoch_iteration : " << (CD_GRAPHSIZE + CD_SET_THREAD - 1) / CD_SET_THREAD << endl;
    while (it < CD_ITERATION)
    {
        if (it % 2 == 0)
        {

            extract_labels<<<init_label_block, init_label_thread>>>(in_out_ptr_gpu, ins_ptr_gpu, outs_ptr_gpu, ins_neighbor_gpu, outs_neighbor_gpu, labels_gpu, global_space_for_label, CD_GRAPHSIZE);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after extract_labels");
            // cout << 5 << endl;
            parallel_sort_labels<<<init_label_block, init_label_thread>>>(in_out_ptr_gpu, global_space_for_label, CD_GRAPHSIZE);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after parallel_sort_labels");
            // cout << 6 << endl;
            LPA<<<init_label_block, init_label_thread>>>(global_space_for_label, in_out_ptr_gpu, labels_gpu, new_labels_gpu, CD_GRAPHSIZE);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after LPA");
        }
        else
        {
            extract_labels<<<init_label_block, init_label_thread>>>(in_out_ptr_gpu, ins_ptr_gpu, outs_ptr_gpu, ins_neighbor_gpu, outs_neighbor_gpu, new_labels_gpu, global_space_for_label, CD_GRAPHSIZE);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after extract_labels");
            parallel_sort_labels<<<init_label_block, init_label_thread>>>(in_out_ptr_gpu, global_space_for_label, CD_GRAPHSIZE);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after parallel_sort_labels");
            LPA<<<init_label_block, init_label_thread>>>(global_space_for_label, in_out_ptr_gpu, new_labels_gpu, labels_gpu, CD_GRAPHSIZE);
            err = hipDeviceSynchronize();
            checkCudaError(err, "hipDeviceSynchronize after LPA");
        }

        it++;
    }

    hipEventRecord(GPUstop, 0);
    hipEventSynchronize(GPUstop);

    hipEventElapsedTime(elapsedTime, GPUstart, GPUstop);
    hipEventDestroy(GPUstart);
    hipEventDestroy(GPUstop);

    ans.resize(CD_GRAPHSIZE);
    hipMemcpy(ans.data(), labels_gpu, CD_GRAPHSIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(outs_ptr_gpu);
    hipFree(ins_ptr_gpu);
    hipFree(labels_gpu);
    hipFree(outs_neighbor_gpu);
    hipFree(ins_neighbor_gpu);
    hipFree(new_labels_gpu);

    return 0;
}
