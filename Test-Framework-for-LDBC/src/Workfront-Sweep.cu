#include "hip/hip_runtime.h"
#include <Workfront-Sweep.cuh>

__device__ __forceinline__ double atomicMinDouble (double * addr, double value) {
    double old;
    old = __longlong_as_double(atomicMin((long long *)addr, __double_as_longlong(value)));
    return old;
}

__global__ void Relax(int* offsets, int* edges, double* weights, double* dis, int* queue, int* queue_size, int* visited) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < *queue_size) {
        int v = queue[idx];

        for (int i = offsets[v]; i < offsets[v + 1]; i++) {
            int new_v = edges[i];
            double weight = weights[i];

            double new_w = dis[v] + weight;

            double old = atomicMinDouble(&dis[new_v], (long long)new_w);

            if (old <= new_w)
				continue;

            atomicExch(&visited[new_v], 1);
        }
    }
}

__global__ void CompactQueue(int V, int* next_queue, int* next_queue_size, int* visited) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < V && visited[idx]) {
        int pos = atomicAdd(next_queue_size, 1);
        next_queue[pos] = idx;
        visited[idx] = 0;
    }
}

void Workfront_Sweep(CSR_graph<double>& input_graph, int source, std::vector<double>& distance, float* elapsedTime ,double max_dis) {
    int V = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
    int E = input_graph.OUTs_Edges.size();

    double* dis;
    int* edges;
    double* weights;
    int* offsets;
    int* visited;
    
    int* queue, * next_queue;
    int* queue_size, * next_queue_size;

    hipMallocManaged((void**)&dis, V * sizeof(double));
    hipMallocManaged((void**)&edges, E * sizeof(int));
    hipMallocManaged((void**)&weights, E * sizeof(double));
    hipMallocManaged((void**)&offsets, (V + 1) * sizeof(int));
    hipMallocManaged((void**)&visited, V * sizeof(int));
    hipMallocManaged((void**)&queue, V * sizeof(int));
    hipMallocManaged((void**)&next_queue, V * sizeof(int));
    hipMallocManaged((void**)&queue_size, sizeof(int));
    hipMallocManaged((void**)&next_queue_size, sizeof(int));

    for (int i = 0; i < V; i++) {
		dis[i] = max_dis;
		visited[i] = 0;
	}
    dis[source] = 0;
    hipMemcpy(offsets, input_graph.OUTs_Neighbor_start_pointers.data(), (V + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(edges, input_graph.OUTs_Edges.data(), E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(weights, input_graph.OUTs_Edge_weights.data(), E * sizeof(double), hipMemcpyHostToDevice);

    *queue_size = 1;
    queue[0] = source;
    *next_queue_size = 0;

    int threadsPerBlock = 1024;
    int numBlocks = 0;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    while (*queue_size > 0) {
		numBlocks = (*queue_size + threadsPerBlock - 1) / threadsPerBlock;
		Relax <<< numBlocks, threadsPerBlock >>> (offsets, edges, weights, dis, queue, queue_size, visited);
		hipDeviceSynchronize();

        hipError_t cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            fprintf(stderr, "Relax kernel launch failed: %s\n", hipGetErrorString(cuda_status));
            return;
        }

		numBlocks = (V + threadsPerBlock - 1) / threadsPerBlock;
		CompactQueue <<< numBlocks, threadsPerBlock >>> (V, next_queue, next_queue_size, visited);
		hipDeviceSynchronize();

        cuda_status = hipGetLastError();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "CompactQueue kernel launch failed: %s\n", hipGetErrorString(cuda_status));
			return;
		}
		
        std::swap(queue, next_queue);

		*queue_size = *next_queue_size;
        *next_queue_size = 0;
	}

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(distance.data(), dis, V * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dis);
    hipFree(edges);
    hipFree(weights);
    hipFree(offsets);
    hipFree(visited);
    hipFree(queue);
    hipFree(next_queue);
    hipFree(queue_size);
    hipFree(next_queue_size);

    return;
}

/*int main()
{
    std::string file_path;
    std::cout << "Please input the file path of the graph: ";
    std::cin >> file_path;
    graph_v_of_v<double> graph;
    graph.txt_read(file_path);
    ARRAY_graph<double> arr_graph = graph.toARRAY();
    float sum = 0;
    int V = arr_graph.Neighbor_start_pointers.size() - 1;
    std::vector<double> distance(V, 0);
    Workfront_Sweep(arr_graph, 0, distance, 1000000);
    for (int i = 0; i < V; i++) {
        Workfront_Sweep(arr_graph, i, distance, 1000000);
        sum += elapsedTime;
        elapsedTime = 0;
    }
    printf("GPU average time: %f ms\n", sum / V);
    return 0;
}*/