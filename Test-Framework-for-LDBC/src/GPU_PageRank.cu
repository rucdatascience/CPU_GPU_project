#include "hip/hip_runtime.h"
#include <GPU_PageRank.cuh>
/* 
Let PRi(v) be the PageRank value of vertex v after iteration i. 
Initially, each vertex v is assigned the same value such that the sum
of all vertex values is 1.
After iteration i, each vertex pushes its PageRank over its outgoing edges to its neighbors.
 The PageRank for each vertex is updated according to the following rule:
PRi(v) = teleport + importance + reredistributed from sinks */
static int ITERATION;
static int ALPHA;
static int GRAPHSIZE;
static int *graphSize;
static int *row_point, *val_col;//in edge pointer,neighbors with in edges
static int *row_size;
static double *row_value;
static vector<int> N_out_zero;//sink vertexs
static int *N_out_zero_gpu;
static int * row_out_ptr;
static vector<double> row_value_vec;
static vector<int> val_col_vec;
static int *verticeOrder;
static int *smallOffset, *normalOffset;
static double *Rank, *diff_array, *reduce_array;
static double *newRank, *F, *temp;
static int out_zero_size;
static double *sink_sum;
void PageRank(graph_structure<double> &graph, float *elapsedTime, vector<double> & result)
{   //allocate GPU memory
    CSR_graph<double> ARRAY_graph = graph.toCSR();
    GRAPHSIZE = ARRAY_graph.OUTs_Neighbor_start_pointers.size() - 1;
    hipMallocManaged(&graphSize, sizeof(int));
    hipMallocManaged(&smallOffset, sizeof(int));
    hipMallocManaged(&normalOffset, sizeof(int));
    hipMallocManaged(&temp, sizeof(double *));
    hipMallocManaged(&row_size, GRAPHSIZE * sizeof(int));
    hipMallocManaged(&row_point, (GRAPHSIZE+1) * sizeof(int));
    hipMallocManaged(&row_out_ptr, (GRAPHSIZE+1) * sizeof(int));
    hipMallocManaged(&sink_sum, sizeof(double));
    hipMallocManaged(&newRank, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&F, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&diff_array, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&reduce_array, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&Rank, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&verticeOrder, GRAPHSIZE * sizeof(int));
    hipMemcpy(row_point, ARRAY_graph.INs_Neighbor_start_pointers.data(), (GRAPHSIZE+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_out_ptr, ARRAY_graph.OUTs_Neighbor_start_pointers.data(), (GRAPHSIZE+1) * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < GRAPHSIZE; i++)
    {
        for (auto it : graph.INs[i])
        {   //Traverse the incoming edges of vertex
            row_value_vec.push_back(1.0 / (graph.OUTs[it.first].size()));//OUTs[it.first].size() is denominator in importance
            val_col_vec.push_back(it.first);//it.first is neighbor
        }
        if(row_out_ptr[i]==row_out_ptr[i+1]){
            //This means that the vertex has no edges
            N_out_zero.push_back(i);
        }
    }
    hipMallocManaged(&N_out_zero_gpu, N_out_zero.size() * sizeof(int));
    hipMemcpy(N_out_zero_gpu, N_out_zero.data(),  N_out_zero.size() * sizeof(int), hipMemcpyHostToDevice);
    out_zero_size=N_out_zero.size();
    ALPHA = graph.pr_damping;//d
    ITERATION = graph.pr_its;
    hipMallocManaged(&row_value, row_value_vec.size() * sizeof(double));
    std::copy(row_value_vec.begin(), row_value_vec.end(), row_value);
    hipMallocManaged(&val_col, val_col_vec.size() * sizeof(int));
    std::copy(val_col_vec.begin(), val_col_vec.end(), val_col);
    dim3 blockPerGrid((GRAPHSIZE + THREAD_PER_BLOCK-1) / THREAD_PER_BLOCK, 1, 1);
    dim3 threadPerGrid(THREAD_PER_BLOCK, 1, 1);

    for (int i = 0; i < GRAPHSIZE; i++)
    {   //Initially, each vertex v is assigned the same value such that the sum of all vertex values is 1.
        Rank[i] = 1.0 / GRAPHSIZE;
    }
    int iteration = 0;
    double d = ALPHA, d_ops = (1 - ALPHA) / GRAPHSIZE;//teleport
    hipEvent_t GPUstart, GPUstop;// record GPU_TIME
    hipEventCreate(&GPUstart);
    hipEventCreate(&GPUstop);
    hipEventRecord(GPUstart, 0);

    while (iteration < ITERATION)
    {
        *sink_sum=0;
        calculate_sink<<<blockPerGrid, threadPerGrid,THREAD_PER_BLOCK*sizeof(double)>>>(Rank, N_out_zero_gpu,out_zero_size,sink_sum);
        hipDeviceSynchronize();
        
        tinySolve<<<blockPerGrid, threadPerGrid>>>(F, Rank, d, row_point, row_size, row_value, val_col, GRAPHSIZE);//importance
        hipDeviceSynchronize();
        //ALPHA/GRAPHSIZE)=d/|v|
        add_scaling<<<blockPerGrid, threadPerGrid>>>(newRank, F, (ALPHA/GRAPHSIZE)*(*sink_sum)+d_ops, GRAPHSIZE);//sum up
        hipDeviceSynchronize();

        temp = newRank;//swap newrank and rank
        newRank = Rank;
        Rank = temp;
        iteration++;
    }
    //get gpu PR algorithm result
    hipMemcpy(result.data(), Rank, GRAPHSIZE * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(GPUstop, 0);
    hipEventSynchronize(GPUstop);

    float CUDAtime = 0;
    hipEventElapsedTime(&CUDAtime, GPUstart, GPUstop);
    *elapsedTime += CUDAtime;
    hipEventDestroy(GPUstart);
    hipEventDestroy(GPUstop);
}


bool cmp(const std::vector<pair<int, int>> &a, const std::vector<pair<int, int>> &b)
{
    return a.size() > b.size();
}

__global__ void add_scaling(double *newRank, double *oldRank, double scaling, int GRAPHSIZE)
{    //Add all the elements in rankvector with teleport and redistributed from sinks
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 0 && tid < GRAPHSIZE)
    {
        newRank[tid] = oldRank[tid] + scaling;
    }
    return;
}

__global__ void tinySolve(double *newRank, double *rank, double scaling, int *row_point, int *row_size, double *row_value, int *val_col, int GRAPHSIZE)
{   //importance
    int tid = blockIdx.x * blockDim.x + threadIdx.x;//tid decides process which vertex
    if (tid >= 0 && tid < GRAPHSIZE)
    {
        int rbegin = row_point[tid];
        int rend = row_point[tid + 1];
        //begin and end of in edges
        double acc = 0;//sum of u belongs to Nin(v)
        for (int c = rbegin; c < rend; c++)
        {   //val_col[c] is neighbor,rank get PR(u) row_value is denominator i.e. Nout
            acc += row_value[c] * (rank[val_col[c]]);
        }
        // printf("tid : %d  acc : %f\n", tid, acc);
        newRank[tid] = acc * scaling;//scaling is damping factor 
    }
    return;
}
__device__ double _atomicAdd(double* address, double val) {
    /* Implementing atomic operations, 
    that is, ensuring that adding operations to a specific
     memory location in a multi-threaded environment are thread safe. */
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void calculate_sink(double* rank, int* N_out_zero_gpu, int out_zero_size, double* sink_sum) {
    //A reduction pattern was used to sum up
    extern __shared__ double sink[];//Declare shared memory
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stid = threadIdx.x;

    if (tid < out_zero_size) {
        sink[stid] = rank[N_out_zero_gpu[tid]];//get PR(w)
    } else {
        sink[stid] = 0;
    }
    __syncthreads();//wait unitl finish Loading data into shared memory
    
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (stid < i) {
            sink[stid] += sink[stid + i];
        }
        __syncthreads();//Synchronize again to ensure that each step of the reduction operation is completed

    }
    if (stid == 0) {
        _atomicAdd(sink_sum, sink[0]);//Write the result of each thread block into the output array
    }
}


// __global__ void vec_diff(double *diff, double *newRank, double *oldRank)
// {

//     __shared__ double s_newRank[512];
//     __shared__ double s_oldRank[512];

//     int idx = threadIdx.x + blockIdx.x * blockDim.x;

//    
//     if (idx < GRAPHSIZE)
//     {

//         s_newRank[threadIdx.x] = newRank[idx];
//         s_oldRank[threadIdx.x] = oldRank[idx];

//         __syncthreads();

//         diff[idx] = abs(s_newRank[threadIdx.x] - s_oldRank[threadIdx.x]);
//     }
// }

// __global__ void reduce_kernel(double *input, double *output)
// {
//     extern __shared__ double sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < GRAPHSIZE)
//         sdata[tid] = input[i];
//     else
//         sdata[tid] = 0.0;
//     __syncthreads();

//     for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
//     {
//         if (tid < s)
//         {
//             // sdata[tid] += sdata[tid + s];
//             sdata[tid] = sdata[tid] > sdata[tid + s] ? sdata[tid] : sdata[tid + s];
//         }
//         __syncthreads();
//     }

//     if (tid == 0)
//         output[blockIdx.x] = sdata[0];
// }

// int main(){
//     std::string file_path;
//     std::cout << "Please input the file path of the graph: ";
//     std::cin >> file_path;
//     graph_structure<double> graph;
//     graph.read_txt(file_path);
//     PageRank(graph);
//     return 0;
// }