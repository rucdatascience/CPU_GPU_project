#include "hip/hip_runtime.h"
#include <GPU_PageRank.cuh>

static int ITERATION;
static int ALPHA;
static int GRAPHSIZE;
static int *graphSize;
static int *row_point, *val_col;
static int *row_size;
static double *row_value;
static vector<int> N_out_zero;
static int *N_out_zero_gpu;
static int * row_out_ptr;
static vector<double> row_value_vec;
static vector<int> val_col_vec;
static int *verticeOrder;
static int *smallOffset, *normalOffset;
static double *Rank, *diff_array, *reduce_array;
static double *newRank, *F, *temp;
static int out_zero_size;
static double *sink_sum;
void PageRank(graph_structure<double> &graph, float *elapsedTime)
{
    CSR_graph<double> ARRAY_graph = graph.toCSR();
    GRAPHSIZE = ARRAY_graph.OUTs_Neighbor_start_pointers.size() - 1;
    hipMallocManaged(&graphSize, sizeof(int));
    hipMallocManaged(&smallOffset, sizeof(int));
    hipMallocManaged(&normalOffset, sizeof(int));
    hipMallocManaged(&temp, sizeof(double *));
    hipMallocManaged(&row_size, GRAPHSIZE * sizeof(int));
    hipMallocManaged(&row_point, (GRAPHSIZE+1) * sizeof(int));
    hipMallocManaged(&row_out_ptr, (GRAPHSIZE+1) * sizeof(int));
    hipMallocManaged(&sink_sum, sizeof(double));
    hipMallocManaged(&newRank, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&F, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&diff_array, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&reduce_array, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&Rank, GRAPHSIZE * sizeof(double));
    hipMallocManaged(&verticeOrder, GRAPHSIZE * sizeof(int));
    hipMemcpy(row_point, ARRAY_graph.INs_Neighbor_start_pointers.data(), (GRAPHSIZE+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_out_ptr, ARRAY_graph.OUTs_Neighbor_start_pointers.data(), (GRAPHSIZE+1) * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < GRAPHSIZE; i++)
    {
        for (auto it : graph.INs[i])
        {
            row_value_vec.push_back(1.0 / (graph.OUTs[it.first].size()));
            val_col_vec.push_back(it.first);
        }
        if(row_out_ptr[i]==row_out_ptr[i+1]){
            N_out_zero.push_back(i);
        }
    }
    hipMallocManaged(&N_out_zero_gpu, N_out_zero.size() * sizeof(int));
    hipMemcpy(N_out_zero_gpu, N_out_zero.data(),  N_out_zero.size() * sizeof(int), hipMemcpyHostToDevice);
    out_zero_size=N_out_zero.size();
    ALPHA = graph.pr_damping;
    ITERATION = graph.pr_its;
    hipMallocManaged(&row_value, row_value_vec.size() * sizeof(double));
    std::copy(row_value_vec.begin(), row_value_vec.end(), row_value);
    hipMallocManaged(&val_col, val_col_vec.size() * sizeof(int));
    std::copy(val_col_vec.begin(), val_col_vec.end(), val_col);
    dim3 blockPerGrid((GRAPHSIZE + THREAD_PER_BLOCK-1) / THREAD_PER_BLOCK, 1, 1);
    dim3 threadPerGrid(THREAD_PER_BLOCK, 1, 1);

    for (int i = 0; i < GRAPHSIZE; i++)
    {
        Rank[i] = 1.0 / GRAPHSIZE;
    }
    int iteration = 0;
    double d = ALPHA, d_ops = (1 - ALPHA) / GRAPHSIZE;
    hipEvent_t GPUstart, GPUstop;
    hipEventCreate(&GPUstart);
    hipEventCreate(&GPUstop);
    hipEventRecord(GPUstart, 0);

    while (iteration < ITERATION)
    {
        *sink_sum=0;
        calculate_sink<<<blockPerGrid, threadPerGrid,THREAD_PER_BLOCK*sizeof(double)>>>(Rank, N_out_zero_gpu,out_zero_size,sink_sum);
        hipDeviceSynchronize();
        
        tinySolve<<<blockPerGrid, threadPerGrid>>>(F, Rank, d, row_point, row_size, row_value, val_col, GRAPHSIZE);
        hipDeviceSynchronize();
        
        add_scaling<<<blockPerGrid, threadPerGrid>>>(newRank, F, (ALPHA/GRAPHSIZE)*(*sink_sum)+d_ops, GRAPHSIZE);
        hipDeviceSynchronize();

        temp = newRank;
        newRank = Rank;
        Rank = temp;
        iteration++;
    }
    hipEventRecord(GPUstop, 0);
    hipEventSynchronize(GPUstop);

    float CUDAtime = 0;
    hipEventElapsedTime(&CUDAtime, GPUstart, GPUstop);
    *elapsedTime += CUDAtime;
    hipEventDestroy(GPUstart);
    hipEventDestroy(GPUstop);
}


bool cmp(const std::vector<pair<int, int>> &a, const std::vector<pair<int, int>> &b)
{
    return a.size() > b.size();
}

__global__ void add_scaling(double *newRank, double *oldRank, double scaling, int GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 0 && tid < GRAPHSIZE)
    {
        newRank[tid] = oldRank[tid] + scaling;
    }
    return;
}

__global__ void tinySolve(double *newRank, double *rank, double scaling, int *row_point, int *row_size, double *row_value, int *val_col, int GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 0 && tid < GRAPHSIZE)
    {
        int rbegin = row_point[tid];
        int rend = row_point[tid + 1];

        double acc = 0;
        for (int c = rbegin; c < rend; c++)
        {
            acc += row_value[c] * (rank[val_col[c]]);
        }
        // printf("tid : %d  acc : %f\n", tid, acc);
        newRank[tid] = acc * scaling;
    }
    return;
}
__device__ double _atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void calculate_sink(double* rank, int* N_out_zero_gpu, int out_zero_size, double* sink_sum) {
    extern __shared__ double sink[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stid = threadIdx.x;

    if (tid < out_zero_size) {
        sink[stid] = rank[N_out_zero_gpu[tid]];
    } else {
        sink[stid] = 0;
    }
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (stid < i) {
            sink[stid] += sink[stid + i];
        }
        __syncthreads();
    }

    if (stid == 0) {
        _atomicAdd(sink_sum, sink[0]);
    }
}


// __global__ void vec_diff(double *diff, double *newRank, double *oldRank)
// {

//     __shared__ double s_newRank[512];
//     __shared__ double s_oldRank[512];

//     int idx = threadIdx.x + blockIdx.x * blockDim.x;

//    
//     if (idx < GRAPHSIZE)
//     {

//         s_newRank[threadIdx.x] = newRank[idx];
//         s_oldRank[threadIdx.x] = oldRank[idx];

//         __syncthreads();

//         diff[idx] = abs(s_newRank[threadIdx.x] - s_oldRank[threadIdx.x]);
//     }
// }

// __global__ void reduce_kernel(double *input, double *output)
// {
//     extern __shared__ double sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < GRAPHSIZE)
//         sdata[tid] = input[i];
//     else
//         sdata[tid] = 0.0;
//     __syncthreads();

//     for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
//     {
//         if (tid < s)
//         {
//             // sdata[tid] += sdata[tid + s];
//             sdata[tid] = sdata[tid] > sdata[tid + s] ? sdata[tid] : sdata[tid + s];
//         }
//         __syncthreads();
//     }

//     if (tid == 0)
//         output[blockIdx.x] = sdata[0];
// }

// int main(){
//     std::string file_path;
//     std::cout << "Please input the file path of the graph: ";
//     std::cin >> file_path;
//     graph_structure<double> graph;
//     graph.read_txt(file_path);
//     PageRank(graph);
//     return 0;
// }